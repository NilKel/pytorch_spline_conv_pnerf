#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t>
__global__ void weighting_bw_w_kernel(
    at::cuda::detail::TensorInfo<scalar_t, int64_t> grad_weight,
    at::cuda::detail::TensorInfo<scalar_t, int64_t> grad_out,
    at::cuda::detail::TensorInfo<scalar_t, int64_t> x,
    at::cuda::detail::TensorInfo<scalar_t, int64_t> basis,
    at::cuda::detail::TensorInfo<int64_t, int64_t> weight_index, size_t numel) {
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (ptrdiff_t i = index; i < numel; i += stride) {
    int64_t e = i / grad_out.sizes[1], m_out = i % grad_out.sizes[1];
    int64_t S = basis.sizes[1], M_in = x.sizes[1], M_out = grad_out.sizes[1];

    auto g =
        grad_out.data[e * grad_out.strides[0] + m_out * grad_out.strides[1]];
    for (ptrdiff_t s = 0; s < S; s++) {
      auto b = basis.data[e * S + s];
      auto wi = weight_index.data[e * S + s];
      for (ptrdiff_t m_in = 0; m_in < M_in; m_in++) {
        auto v = g * b * x.data[e * x.strides[0] + m_in * x.strides[1]];
        atomicAdd(&grad_weight.data[wi * M_in * M_out + m_in * M_out + m_out],
                  v);
      }
    }
  }
}

at::Tensor weighting_bw_w_cuda(at::Tensor grad_out, at::Tensor x,
                               at::Tensor basis, at::Tensor weight_index,
                               int64_t K) {
  hipSetDevice(grad_out.get_device());
  auto M_in = x.size(1), M_out = grad_out.size(1);
  auto grad_weight = at::zeros({K, M_in, M_out}, grad_out.options());
  AT_DISPATCH_FLOATING_TYPES(grad_out.scalar_type(), "weighting_bw_w", [&] {
    weighting_bw_w_kernel<scalar_t><<<BLOCKS(grad_out.numel()), THREADS>>>(
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(grad_weight),
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(grad_out),
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(x),
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(basis),
        at::cuda::detail::getTensorInfo<int64_t, int64_t>(weight_index),
        grad_out.numel());
  });
  return grad_weight;
}

template <typename scalar_t>
__global__ void weighting_bw_b_kernel(
    at::cuda::detail::TensorInfo<scalar_t, int64_t> grad_basis,
    at::cuda::detail::TensorInfo<scalar_t, int64_t> grad_out,
    at::cuda::detail::TensorInfo<scalar_t, int64_t> x,
    at::cuda::detail::TensorInfo<scalar_t, int64_t> weight,
    at::cuda::detail::TensorInfo<int64_t, int64_t> weight_index, size_t numel) {
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (ptrdiff_t i = index; i < numel; i += stride) {
    int64_t e = i / grad_out.sizes[1], m_out = i % grad_out.sizes[1];
    auto S = grad_basis.sizes[1];

    auto g =
        grad_out.data[e * grad_out.strides[0] + m_out * grad_out.strides[1]];
    for (ptrdiff_t s = 0; s < S; s++) {
      scalar_t v = 0;
      auto wi = weight_index.data[e * S + s];
      for (ptrdiff_t m_in = 0; m_in < x.sizes[1]; m_in++) {
        auto w = weight.data[wi * weight.strides[0] + m_in * weight.strides[1] +
                             m_out * weight.strides[2]];
        v += g * w * x.data[e * x.strides[0] + m_in * x.strides[1]];
      }
      atomicAdd(&grad_basis.data[e * S + s], v);
    }
  }
}

at::Tensor weighting_bw_b_cuda(at::Tensor grad_out, at::Tensor x,
                               at::Tensor weight, at::Tensor weight_index) {
  hipSetDevice(grad_out.get_device());
  auto E = x.size(0), S = weight_index.size(1);
  auto grad_basis = at::zeros({E, S}, grad_out.options());
  AT_DISPATCH_FLOATING_TYPES(grad_out.scalar_type(), "weighting_bw_b", [&] {
    weighting_bw_b_kernel<scalar_t><<<BLOCKS(grad_out.numel()), THREADS>>>(
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(grad_basis),
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(grad_out),
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(x),
        at::cuda::detail::getTensorInfo<scalar_t, int64_t>(weight),
        at::cuda::detail::getTensorInfo<int64_t, int64_t>(weight_index),
        grad_out.numel());
  });
  return grad_basis;
}
