#include "hip/hip_runtime.h"
#include "multispline_basis_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t, int64_t degree> struct Basis {
  static inline __device__ scalar_t forward(scalar_t v, int64_t k_mod) {
    if (degree == 1) {
      return 1. - v - k_mod + 2. * v * k_mod;
    } else if (degree == 2) {
      if (k_mod == 0)
        return 0.5 * v * v - v + 0.5;
      else if (k_mod == 1)
        return -v * v + v + 0.5;
      else
        return 0.5 * v * v;
    } else if (degree == 3) {
      if (k_mod == 0)
        return (1. - v) * (1. - v) * (1. - v) / 6.;
      else if (k_mod == 1)
        return (3. * v * v * v - 6. * v * v + 4.) / 6.;
      else if (k_mod == 2)
        return (-3. * v * v * v + 3. * v * v + 3. * v + 1.) / 6.;
      else
        return v * v * v / 6.;
    } else {
      return (scalar_t)-1.;
    }
  }

  static inline __device__ scalar_t backward(scalar_t v, int64_t k_mod) {
    if (degree == 1) {
      return 2 * k_mod - 1;
    } else if (degree == 2) {
      if (k_mod == 0)
        return v - 1.;
      else if (k_mod == 1)
        return -2. * v + 1.;
      else
        return v;
    } else if (degree == 3) {
      if (k_mod == 0)
        return (-v * v + 2. * v - 1.) / 2.;
      else if (k_mod == 1)
        return (3. * v * v - 4. * v) / 2.;
      else if (k_mod == 2)
        return (-3. * v * v + 2. * v + 1.) / 2.;
      else
        return v * v / 2.;
    } else {
      return (scalar_t)-1.;
    }
  }
};


template <typename scalar_t, int64_t degree>
__global__ void
multispline_basis_fw_kernel(const scalar_t *pseudo, const int64_t *kernel_size,
                       const uint8_t *is_open_spline, scalar_t *basis,
                       int64_t *weight_index, int64_t E, int64_t D, int64_t S,
                       int64_t numel, int64_t levels, const int64_t *resolution, int64_t log2_hashmap_size, int64_t cellsize,
                        const scalar_t *xyz, const int64_t *point_index, const int64_t *primes, const int64_t *offsets) {
  
  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / S;
  const int64_t s = thread_idx % S;
  if (thread_idx < numel) {
    
    unsigned long point_idx = static_cast<unsigned long>(point_index[e]);
    
    // unsigned int primes[16] = {0, 0, 0, 0, 0, 0, 715902911, 729070343, 919548613, 449542579, 530997011, 314167211, 146148241, 458291711, 225061747, 385280261};
    // unsigned int offsets[16] = {8, 27, 64, 125, 216, 343, 512,729, 1000,1331,1728,2197,2744,3375,4096,4913 };
    
    #pragma unroll(16)
    for(int64_t level = 0; level<levels; level++){
      int64_t k = s, wi = 0, wi_offset = 1;
      scalar_t b = (scalar_t)1.;
      #pragma unroll(3)
      for (int64_t d = 0; d < D; d++) {

        
        const int64_t k_mod = k % (degree + 1);
        k /= degree + 1;
        scalar_t v;
       
        v = pseudo[e * D * levels + d*levels + level];
        // array of wi here and array of offsets used
        // wi += (((int64_t)v + k_mod) % kernel_size[level*D+d]) * wi_offset;
        wi = wi ^ ((((int64_t)v + k_mod) % kernel_size[level*D+d])*primes[d]);
        wi_offset *= kernel_size[level*D+d];

        v -= floor(v);
        v = Basis<scalar_t, degree>::forward(v, k_mod);
        b *= v;
      }
      
      // unsigned int temp = ((wi + point_idx *offsets[level]) *(primes[level])& ((1 << log2_hashmap_size) - 1));
      unsigned int temp = (wi ^ point_idx*primes[4]) & ((1 << log2_hashmap_size) - 1);
      unsigned int temp = (wi ^ point_idx) & ((1 << log2_hashmap_size) - 1);
      // unsigned int temp = (((wi + point_idx *offsets[level])) ^ primes[level]);
      // // temp = temp % 4294967295;
      // hashed_coords = (temp ) & ((1 << log2_hashmap_size) - 1);
      
      basis[e*S*levels  + s * levels + level] = b;
      weight_index[e*S*levels  + s * levels + level] = temp;
    }
  }

}


// template <typename scalar_t, int64_t degree>
// __global__ void
// multispline_basis_fw_kernel(const scalar_t *pseudo, const int64_t *kernel_size,
//                        const uint8_t *is_open_spline, scalar_t *basis,
//                        int64_t *weight_index, int64_t E, int64_t D, int64_t S,
//                        int64_t numel, int64_t levels, const int64_t *resolution, int64_t log2_hashmap_size, int64_t cellsize,
//                         const scalar_t *xyz, const int64_t *point_index, const int64_t *primes, const int64_t *offsets) {
  
//   const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
//   const int64_t e = thread_idx / S;
//   const int64_t s = thread_idx % S;
//   if (thread_idx < numel) {
    
    
   
    
//     int64_t k = s, wi_offset = 1;
//       // wi and wi offset are now arrays. so is basis.
//     #pragma unroll(3)
//     for (int64_t d = 0; d < D; d++) {

      
//       const int64_t k_mod = k % (degree + 1);
//       k /= degree + 1;
//       #pragma unroll(16)
//       for(int64_t level = 0; level<levels; level++){
//         scalar_t v;

//         v = pseudo[e * D * levels + d*levels + level];
        
//         // array of wi here and array of offsets used
//         weight_index[e*S*levels  + s * levels + level] += (((int64_t)v + k_mod) % kernel_size[level*D+d]) * wi_offset;
//         wi_offset *= kernel_size[level*D+d];

//         v -= floor(v);
//         v = Basis<scalar_t, degree>::forward(v, k_mod);
//         basis[e*S*levels  + s * levels + level] *= v;
//       }
     
//     }
//   }

// }


// template <typename scalar_t, int64_t degree>
// __global__ void
// multispline_basis_fw_kernel(const scalar_t *pseudo, const int64_t *kernel_size,
//                        const uint8_t *is_open_spline, scalar_t *basis,
//                        int64_t *weight_index, int64_t E, int64_t D, int64_t S,
//                        int64_t numel, int64_t levels, const int64_t *resolution, int64_t log2_hashmap_size, int64_t cellsize,
//                         const scalar_t *xyz, const int64_t *point_index, const int64_t *primes, const int64_t *offsets, const int64_t *kernel_power) {
  
//   const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
//   const int64_t e = thread_idx / S;
//   const int64_t s = thread_idx % S;
//   if (thread_idx < numel) {
    
//     unsigned long point_idx = static_cast<unsigned long>(point_index[e]);

//     unsigned int hashed_coords = 0;
   
    
//     int64_t k = s, wi_offset_power = 0;
//       // wi and wi offset are now arrays. so is basis.
//     #pragma unroll(3)
//     for (int64_t d = 0; d < D; d++) {

      
//       const int64_t k_mod = k & (degree);
//       k = k >> 1;
//       #pragma unroll(16)
//       for(int64_t level = 0; level<levels; level++){
//         scalar_t v;

//         v = pseudo[e * D * levels + d*levels + level];
        
//         // array of wi here and array of offsets used
//         weight_index[e*S*levels  + s * levels + level] += (((int64_t)v + k_mod) & (kernel_size[level]-1)) << wi_offset_power;
//         wi_offset_power += kernel_power[level];

//         v -= floor(v);
//         v = Basis<scalar_t, degree>::forward(v, k_mod);
//         basis[e*S*levels  + s * levels + level] *= v;
//       }
     
//     }
//   }

// }


std::tuple<torch::Tensor, torch::Tensor>
multispline_basis_fw_cuda(torch::Tensor pseudo, torch::Tensor kernel_size,
                     torch::Tensor is_open_spline, int64_t degree, torch::Tensor resolution, int64_t log2_hashmap_size, int64_t cellsize,
                    torch::Tensor xyz, torch::Tensor point_index, torch::Tensor primes, torch::Tensor offsets) {
  CHECK_CUDA(pseudo);
  CHECK_CUDA(kernel_size);
  CHECK_CUDA(is_open_spline);
  hipSetDevice(pseudo.get_device());

  // modulo condition on pseudo and kernel size
  CHECK_INPUT(is_open_spline.dim());
  CHECK_INPUT(pseudo.size(1) == is_open_spline.numel());
  
  int64_t E = pseudo.size(0);
  auto D = pseudo.size(1);
  auto levels = resolution.size(0);
  int64_t S = ((int64_t)(powf(degree + 1, D) + 0.5));
  auto primes_data = primes.data_ptr<int64_t>();
  auto offsets_data = offsets.data_ptr<int64_t>();
  auto basis = at::ones({E, S,levels}, pseudo.options());
  auto weight_index = at::zeros({E, S, levels}, kernel_size.options());
  auto resolution_data = resolution.data_ptr<int64_t>();
  auto kernel_size_data = kernel_size.data_ptr<int64_t>();
  // auto kernel_power_data = kernel_power.data_ptr<int64_t>();
  auto is_open_spline_data = is_open_spline.data_ptr<uint8_t>();
  auto weight_index_data = weight_index.data_ptr<int64_t>();
  auto point_index_data = point_index.data_ptr<int64_t>();
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(pseudo.scalar_type(), "basis_fw", [&] {
    auto pseudo_data = pseudo.data_ptr<scalar_t>();
    auto basis_data = basis.data_ptr<scalar_t>();
    auto xyz_data = xyz.data_ptr<scalar_t>();
    AT_DISPATCH_DEGREE_TYPES(degree, [&] {
      multispline_basis_fw_kernel<scalar_t, DEGREE>
          <<<BLOCKS(E*S), THREADS, 0, stream>>>(
              pseudo_data, kernel_size_data, is_open_spline_data, basis_data,
              weight_index_data, E, D, S, E*S,levels, resolution_data, log2_hashmap_size, cellsize, xyz_data, point_index_data, primes_data, offsets_data);
    });
  });

  return std::make_tuple(basis, weight_index);
}

template <typename scalar_t, int64_t degree>
__global__ void
multispline_basis_bw_kernel(const scalar_t *grad_basis, const scalar_t *pseudo,
                       const int64_t *kernel_size,
                       const uint8_t *is_open_spline, scalar_t *grad_pseudo,
                       int64_t E, int64_t D, int64_t S, int64_t numel, int64_t levels) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / D;
  const int64_t d = thread_idx % D;

  if (thread_idx < numel) {
    scalar_t g = (scalar_t)0., tmp;
    for(int64_t level = 0; level < levels; level++){
        for (ptrdiff_t s = 0; s < S; s++) {
            int64_t k_mod = (s / (int64_t)(powf(degree + 1, d) + 0.5)) % (degree + 1);
            
            scalar_t v;
            v = pseudo[e * D * levels + d*levels + level];
            v -= floor(v);
            v = Basis<scalar_t, degree>::backward(v, k_mod);
            tmp = v;

            for (int64_t d_it = 1; d_it < D; d_it++) {
                const int64_t d_new = d_it - (d >= d_it);
                k_mod = (s / (int64_t)(powf(degree + 1, d_new) + 0.5)) % (degree + 1);
                v = pseudo[e * D * levels + d*levels + level];
                v -= floor(v);
                v = Basis<scalar_t, degree>::forward(v, k_mod);
                tmp *= v;
            }
            g += tmp * grad_basis[e * S * levels + s*levels +level];
            }
            g *= kernel_size[level*D+d] - degree * is_open_spline[d];
            grad_pseudo[thread_idx] = g;
        }
  }
}

torch::Tensor multispline_basis_bw_cuda(torch::Tensor grad_basis,
                                   torch::Tensor pseudo,
                                   torch::Tensor kernel_size,
                                   torch::Tensor is_open_spline,
                                   int64_t degree) {
  CHECK_CUDA(grad_basis);
  CHECK_CUDA(pseudo);
  CHECK_CUDA(kernel_size);
  CHECK_CUDA(is_open_spline);
  hipSetDevice(grad_basis.get_device());

  CHECK_INPUT(grad_basis.size(0) == pseudo.size(0));
  CHECK_INPUT(is_open_spline.dim());
  CHECK_INPUT(pseudo.size(1) == is_open_spline.numel());


  auto E = pseudo.size(0);
  auto D = pseudo.size(1);
  auto S = grad_basis.size(1);
  auto levels = kernel_size.size(0);

  auto grad_pseudo = at::empty({E, D}, pseudo.options());

  auto kernel_size_data = kernel_size.data_ptr<int64_t>();
  auto is_open_spline_data = is_open_spline.data_ptr<uint8_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(pseudo.scalar_type(), "basis_bw", [&] {
    auto grad_basis_data = grad_basis.data_ptr<scalar_t>();
    auto pseudo_data = pseudo.data_ptr<scalar_t>();
    auto grad_pseudo_data = grad_pseudo.data_ptr<scalar_t>();

    AT_DISPATCH_DEGREE_TYPES(degree, [&] {
      multispline_basis_bw_kernel<scalar_t, DEGREE>
          <<<BLOCKS(grad_pseudo.numel()), THREADS, 0, stream>>>(
              grad_basis_data, pseudo_data, kernel_size_data,
              is_open_spline_data, grad_pseudo_data, E, D, S,
              grad_pseudo.numel(),levels);
    });
  });

  return grad_pseudo;
}
